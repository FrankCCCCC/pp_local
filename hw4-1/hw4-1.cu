#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define SIZEOFINT sizeof(int)
const int INF = ((1 << 30) - 1);
const int blockdim_x = 8, blockdim_y = 64;
// const int blockdim_x = 2, blockdim_y = 2;
const dim3 block_dim(blockdim_x, blockdim_y);
const int B = 64;
// const int B = 2;
const int Share_Mem_Size = 64;
const int Share_Mem_Row_Size = B;
int n, m;
int *Dist;
int *Dist_cuda;

void show_mat(int *start_p, int vertex_num){
    for(int i = 0; i < vertex_num; i++){
        for(int j = 0; j < vertex_num; j++){
            if(start_p[i * vertex_num + j] == INF){
                printf("INF\t  ");
            }else{
                printf("%d\t  ", start_p[i * vertex_num + j]);
            }
            
        }
        printf("\n");
    }
}

void malloc_Dist(int vertex_num){Dist = (int*)malloc(SIZEOFINT * vertex_num * vertex_num);}
int getDist(int i, int j, int vertex_num){return Dist[i * vertex_num + j];}
int *getDistAddr(int i, int j, int vertex_num){return &(Dist[i * vertex_num + j]);}
void setDist(int i, int j, int val, int vertex_num){Dist[i * vertex_num + j] = val;}

void setup_DistCuda(int vertex_num){
    hipMalloc((void **)&Dist_cuda, SIZEOFINT * vertex_num * vertex_num);
    hipMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), hipMemcpyHostToDevice);
}
void back_DistCuda(int vertex_num){
    hipMemcpy(Dist, Dist_cuda, (n * n * SIZEOFINT), hipMemcpyDeviceToHost);
}
// int getDistCuda(int i, int j, int vertex_num){return Dist_cuda[i * vertex_num + j];}
// int *getDistAddrCuda(int i, int j, int vertex_num){return &(Dist_cuda[i * vertex_num + j]);}
// void setDistCuda(int i, int j, int val, int vertex_num){Dist_cuda[i * vertex_num + j] = val;}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    malloc_Dist(n);
    // malloc_DistCuda(n);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                setDist(i, j, 0, n);
                // Dist[i][j] = 0;
            } else {
                setDist(i, j, INF, n);
                // Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; i++) {
        fread(pair, sizeof(int), 3, file);
        setDist(pair[0], pair[1], pair[2], n);
        // Dist[pair[0]][pair[1]] = pair[2];
    }
    // cudaMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), cudaMemcpyHostToDevice);
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (Dist[i][j] >= INF) Dist[i][j] = INF;
            if (getDist(i, j, n) >= INF) setDist(i, j, INF, n);
        }
        // fwrite(Dist[i], sizeof(int), n, outfile);
        // fwrite(getDistAddr(i, 0, n), sizeof(int), n, outfile);
    }
    fwrite(getDistAddr(0, 0, n), sizeof(int), n * n, outfile);
    fclose(outfile);
}

__device__ void assignAij(int *dist, int a[Share_Mem_Size * Share_Mem_Size], int vertex_num, int Round, int block_internal_start_x, int block_internal_end_x, int block_internal_start_y, int block_internal_end_y){
    for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
        for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
            a[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)] = dist[i * vertex_num + j];
        }
    }
}

__device__ void assignCkj(int *dist, int c[Share_Mem_Size * Share_Mem_Size], int vertex_num, int Round, int block_internal_start_x, int block_internal_end_x, int block_internal_start_y, int block_internal_end_y){
    for (int k = Round * B + threadIdx.x; k < (Round + 1) * B && k < vertex_num; k+=blockDim.x) {
        for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
            c[(k - Round * B) * Share_Mem_Row_Size + (j - block_internal_start_y)] = dist[k * vertex_num + j];
        }
    }
}

__device__ void assignBik(int *dist, int b[Share_Mem_Size * Share_Mem_Size], int vertex_num, int Round, int block_internal_start_x, int block_internal_end_x, int block_internal_start_y, int block_internal_end_y){
    for (int k = Round * B + threadIdx.y; k < (Round + 1) * B && k < vertex_num; k+=blockDim.y) {
        for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
            b[(i - block_internal_start_x) * Share_Mem_Row_Size + (k - Round * B)] = dist[i * vertex_num + k];
        }
    }
}

__device__ void assignBik_r(int *dist, int b[Share_Mem_Size * Share_Mem_Size], int vertex_num, int Round, int block_internal_start_x, int block_internal_end_x, int block_internal_start_y, int block_internal_end_y){
    for (int k = Round * B + threadIdx.y; k < (Round + 1) * B && k < vertex_num; k+=blockDim.y) {
        for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
            b[(k - Round * B) * Share_Mem_Row_Size + (i - block_internal_start_x)] = dist[i * vertex_num + k];
        }
    }
}

__device__ void relax(int a[Share_Mem_Size * Share_Mem_Size], int b[Share_Mem_Size * Share_Mem_Size], int c[Share_Mem_Size * Share_Mem_Size], int vertex_num, int Round, int block_internal_start_x, int block_internal_end_x, int block_internal_start_y, int block_internal_end_y){
    // Relax Path
    for (int k = Round * B; k < (Round + 1) * B && k < vertex_num; k++) {
        for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
            int bv = b[(i - block_internal_start_x) * Share_Mem_Row_Size + (k - Round * B)];
            for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
                int d = bv + c[(k - Round * B) * Share_Mem_Row_Size + (j - block_internal_start_y)];
                if (d < a[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)]) {
                    a[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)] = d;
                }
            }
        }
        __syncthreads();
    }
}

__device__ void relax_r(int a[Share_Mem_Size * Share_Mem_Size], int b[Share_Mem_Size * Share_Mem_Size], int c[Share_Mem_Size * Share_Mem_Size], int vertex_num, int Round, int block_internal_start_x, int block_internal_end_x, int block_internal_start_y, int block_internal_end_y){
    // Relax Path
    for (int k = Round * B; k < (Round + 1) * B && k < vertex_num; k++) {
        for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
            int bv = b[(k - Round * B) * Share_Mem_Row_Size + (i - block_internal_start_x)];
            for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
                int d = bv + c[(k - Round * B) * Share_Mem_Row_Size + (j - block_internal_start_y)];
                if (d < a[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)]) {
                    a[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)] = d;
                }
            }
        }
        __syncthreads();
    }
    // __syncthreads();
}

__device__ void flush(int *dist, int a[Share_Mem_Size * Share_Mem_Size], int vertex_num, int block_internal_start_x, int block_internal_end_x, int block_internal_start_y, int block_internal_end_y){
    // Move modified block to global memory
    for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
        for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
            dist[i * vertex_num + j] = a[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)];
        }
    }
}
__global__ void phase1_cal_cuda(int *dist, int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_height;
    int block_end_y = block_start_y + block_width;
    // printf("%d\n", dist[1]);
    // i-j block
    __shared__ int a[Share_Mem_Size * Share_Mem_Size];
    // i-k block
    __shared__ int b[Share_Mem_Size * Share_Mem_Size];
    // k-j block
    __shared__ int c[Share_Mem_Size * Share_Mem_Size];

    for (int b_i = block_start_x + blockIdx.x; b_i < block_end_x; b_i+=gridDim.x) {
        for (int b_j = block_start_y + blockIdx.y; b_j < block_end_y; b_j+=gridDim.y) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times

            // To calculate original index of elements in the block (b_i, b_j)
            // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;

            if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
            if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;
            
            assignAij(dist, a, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // assignCkj(dist, c, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Reverse the row and column to ensure column-major iteration
            // assignBik_r(dist, b, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            __syncthreads();

            // Relax Path
            relax(a, a, a, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Move modified block to global memory
            flush(dist, a, vertex_num, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
        }
    }
}
__global__ void phase3_cal_cuda(int *dist, int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_height;
    int block_end_y = block_start_y + block_width;
    // printf("%d\n", dist[1]);
    // i-j block
    __shared__ int a[Share_Mem_Size * Share_Mem_Size];
    // i-k block
    __shared__ int b[Share_Mem_Size * Share_Mem_Size];
    // k-j block
    __shared__ int c[Share_Mem_Size * Share_Mem_Size];

    for (int b_i = block_start_x + blockIdx.x; b_i < block_end_x; b_i+=gridDim.x) {
        for (int b_j = block_start_y + blockIdx.y; b_j < block_end_y; b_j+=gridDim.y) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times

            // To calculate original index of elements in the block (b_i, b_j)
            // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;

            if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
            if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;
            
            assignAij(dist, a, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            assignCkj(dist, c, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Reverse the row and column to ensure column-major iteration
            assignBik_r(dist, b, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // assignBik(dist, b, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            __syncthreads();

            // Relax Path
            relax_r(a, b, c, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // relax(a, b, c, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Move modified block to global memory
            flush(dist, a, vertex_num, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
        }
    }
}
__global__ void phase21_cal_cuda(int *dist, int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_height;
    int block_end_y = block_start_y + block_width;
    // printf("%d\n", dist[1]);
    // i-j block
    __shared__ int a[Share_Mem_Size * Share_Mem_Size];
    // i-k block
    __shared__ int b[Share_Mem_Size * Share_Mem_Size];
    // k-j block
    __shared__ int c[Share_Mem_Size * Share_Mem_Size];

    for (int b_i = block_start_x + blockIdx.x; b_i < block_end_x; b_i+=gridDim.x) {
        for (int b_j = block_start_y + blockIdx.y; b_j < block_end_y; b_j+=gridDim.y) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times

            // To calculate original index of elements in the block (b_i, b_j)
            // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;

            if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
            if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;
            
            assignAij(dist, a, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // assignCkj(dist, c, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Reverse the row and column to ensure column-major iteration
            assignBik_r(dist, b, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            __syncthreads();

            // Relax Path
            relax_r(a, b, a, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Move modified block to global memory
            flush(dist, a, vertex_num, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
        }
    }
}
__global__ void phase22_cal_cuda(int *dist, int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_height;
    int block_end_y = block_start_y + block_width;
    // printf("%d\n", dist[1]);
    // i-j block
    __shared__ int a[Share_Mem_Size * Share_Mem_Size];
    // i-k block
    __shared__ int b[Share_Mem_Size * Share_Mem_Size];
    // k-j block
    __shared__ int c[Share_Mem_Size * Share_Mem_Size];

    for (int b_i = block_start_x + blockIdx.x; b_i < block_end_x; b_i+=gridDim.x) {
        for (int b_j = block_start_y + blockIdx.y; b_j < block_end_y; b_j+=gridDim.y) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times

            // To calculate original index of elements in the block (b_i, b_j)
            // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;

            if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
            if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;
            
            assignAij(dist, a, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            assignCkj(dist, c, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Reverse the row and column to ensure column-major iteration
            // assignBik_r(dist, b, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            __syncthreads();

            // Relax Path
            relax(a, a, c, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
            // Move modified block to global memory
            flush(dist, a, vertex_num, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
        }
    }
}
// __global__ void cal_cuda(int *dist, int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
//     int block_end_x = block_start_x + block_height;
//     int block_end_y = block_start_y + block_width;
//     // printf("%d\n", dist[1]);
//     // i-j block
//     int (*AM)[Share_Mem_Size * Share_Mem_Size];
//     __shared__ int a[Share_Mem_Size * Share_Mem_Size];
//     // i-k block
//     int (*BM)[Share_Mem_Size * Share_Mem_Size];
//     __shared__ int b[Share_Mem_Size * Share_Mem_Size];
//     // k-j block
//     int (*CM)[Share_Mem_Size * Share_Mem_Size];
//     __shared__ int c[Share_Mem_Size * Share_Mem_Size];

//     for (int b_i = block_start_x + blockIdx.x; b_i < block_end_x; b_i+=gridDim.x) {
//         for (int b_j = block_start_y + blockIdx.y; b_j < block_end_y; b_j+=gridDim.y) {
//             // To calculate B*B elements in the block (b_i, b_j)
//             // For each block, it need to compute B times

//             // To calculate original index of elements in the block (b_i, b_j)
//             // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
//             char is_reverse = 0;
//             int block_internal_start_x = b_i * B;
//             int block_internal_end_x = (b_i + 1) * B;
//             int block_internal_start_y = b_j * B;
//             int block_internal_end_y = (b_j + 1) * B;

//             if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
//             if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;
            
//             // if(threadIdx.x == 0 && threadIdx.y == 0){
//             //     printf("(%d %d) A(%d:%d, %d:%d) B(%d:%d, %d:%d) C(%d:%d, %d:%d) CAL(%d:%d, %d:%d, %d:%d)\n", 
//             //            blockDim.x, blockDim.y, 
//             //            block_internal_start_x + threadIdx.x, block_internal_end_x, block_internal_start_y + threadIdx.y, block_internal_end_y,
//             //            block_internal_start_x + threadIdx.x, block_internal_end_x, Round * B, (Round + 1) * B < vertex_num? (Round + 1) * B : vertex_num,
//             //            Round * B, (Round + 1) * B < vertex_num? (Round + 1) * B : vertex_num, block_internal_start_y + threadIdx.y, block_internal_end_y,
//             //            block_internal_start_x + threadIdx.x, block_internal_end_x, block_internal_start_y + threadIdx.y, block_internal_end_y, Round * B, (Round + 1) * B < vertex_num? (Round + 1) * B : vertex_num
//             //         );
//             // }
            
//             AM = &a;
//             for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
//                 for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
//                     a[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)] = dist[i * vertex_num + j];
//                 }
//             }

//             if(Round != b_i){
//                 CM = &c;
//                 for (int k = Round * B + threadIdx.x; k < (Round + 1) * B && k < vertex_num; k+=blockDim.x) {
//                     for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
//                         c[(k - Round * B) * Share_Mem_Row_Size + (j - block_internal_start_y)] = dist[k * vertex_num + j];
//                     }
//                 }
//             }else{CM = &a;}

//             if(Round != b_j){
//                 BM = &b;
//                 is_reverse = 1;
//                 for (int k = Round * B + threadIdx.y; k < (Round + 1) * B && k < vertex_num; k+=blockDim.y) {
//                     for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
//                         // b[(i - block_internal_start_x) * Share_Mem_Row_Size + (k - Round * B)] = dist[i * vertex_num + k];
//                         b[(k - Round * B) * Share_Mem_Row_Size + (i - block_internal_start_x)] = dist[i * vertex_num + k];
//                     }
//                 }
//             }else{BM = &a;}
//             __syncthreads();

//             // Relax Path
//             for (int k = Round * B; k < (Round + 1) * B && k < vertex_num; k++) {
//                 for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
//                     int bv = 0;
//                     // bv = (*BM)[(i - block_internal_start_x) * Share_Mem_Row_Size + (k - Round * B)];

//                     if(is_reverse){bv = (*BM)[(k - Round * B) * Share_Mem_Row_Size + (i - block_internal_start_x)];}
//                     else{bv = (*BM)[(i - block_internal_start_x) * Share_Mem_Row_Size + (k - Round * B)];}

//                     for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
//                         int d = bv + (*CM)[(k - Round * B) * Share_Mem_Row_Size + (j - block_internal_start_y)];
//                         if (d < (*AM)[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)]) {
//                             (*AM)[(i - block_internal_start_x) * Share_Mem_Row_Size + (j - block_internal_start_y)] = d;
//                         }
//                     }
//                 }
//                 __syncthreads();
//             }
//             // relax(AM, BM, CM, vertex_num, Round, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
//             // Move modified block to global memory
//             flush(dist, (*AM), vertex_num, block_internal_start_x, block_internal_end_x, block_internal_start_y, block_internal_end_y);
//         }
//     }
// }

void block_FW_cuda(int B) {
    int round = (n + B - 1) / B;
    for (int r = 0; r < round; r++) {
        // printf("Round: %d in total: %d\n", r, round);
        fflush(stdout);
        /* Phase 1*/
        phase1_cal_cuda<<<1, block_dim>>>(Dist_cuda, n, m, B, r, r, r, 1, 1);

        /* Phase 2*/
        const int num_stream = 2;
        hipStream_t streams[num_stream];
        for(int i=0; i<num_stream; i++) {hipStreamCreate(&streams[i]);}
        phase21_cal_cuda<<<round, block_dim, 0>>>(Dist_cuda, n, m, B, r, r, 0, round, 1);
        phase22_cal_cuda<<<round, block_dim, 1>>>(Dist_cuda, n, m, B, r, 0, r, 1, round);
        // cudaDeviceSynchronize();
        for(int i=0; i<num_stream; i++) {
            hipStreamDestroy(streams[i]);
        }

        // printf("After\n");
        /* Phase 3*/
        // const dim3 grid_dim0(r, r);
        // const dim3 grid_dim1(round - r - 1, r);
        // const dim3 grid_dim2(r, round - r - 1);
        // const dim3 grid_dim3(round - r - 1, round - r - 1);
        // const int num_stream3 = 2;
        // cudaStream_t streams3[num_stream];
        // for(int i=0; i<num_stream3; i++) {cudaStreamCreate(&streams3[i]);}
        // phase3_cal_cuda<<<grid_dim0, block_dim, 0>>>(Dist_cuda, n, m, B, r, 0, 0, r, r);
        // phase3_cal_cuda<<<grid_dim1, block_dim, 1>>>(Dist_cuda, n, m, B, r, 0, r + 1, round - r - 1, r);
        // phase3_cal_cuda<<<grid_dim2, block_dim, 0>>>(Dist_cuda, n, m, B, r, r + 1, 0, r, round - r - 1);
        // phase3_cal_cuda<<<grid_dim3, block_dim, 1>>>(Dist_cuda, n, m, B, r, r + 1, r + 1, round - r - 1, round - r - 1);
        // for(int i=0; i<num_stream3; i++) {cudaStreamDestroy(streams3[i]);}

        const dim3 grid_dim(round, round);
        phase3_cal_cuda<<<grid_dim, block_dim>>>(Dist_cuda, n, m, B, r, 0, 0, round, round);

        // const dim3 grid_dim0(r, r);
        // const dim3 grid_dim1(round - r - 1, r);
        // const dim3 grid_dim2(r, round - r - 1);
        // const dim3 grid_dim3(round - r - 1, round - r - 1);
        // cal_cuda<<<grid_dim0, block_dim>>>(Dist_cuda, n, m, B, r, 0, 0, r, r);
        // cal_cuda<<<grid_dim1, block_dim>>>(Dist_cuda, n, m, B, r, 0, r + 1, round - r - 1, r);
        // cal_cuda<<<grid_dim2, block_dim>>>(Dist_cuda, n, m, B, r, r + 1, 0, r, round - r - 1);
        // cal_cuda<<<grid_dim3, block_dim>>>(Dist_cuda, n, m, B, r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    // show_mat(getDistAddr(0, 0, n), n);
    setup_DistCuda(n);
    // printf("Vertice: %d, Edge: %d, B: %d\n", n, m, B);
    block_FW_cuda(B);
    back_DistCuda(n);
    // show_mat(getDistAddr(0, 0, n), n);
    
    output(argv[2]);
    // show_mat(getDistAddr(0, 0, n), n);
    return 0;
}